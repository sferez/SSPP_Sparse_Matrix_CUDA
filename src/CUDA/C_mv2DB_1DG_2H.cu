#include "hip/hip_runtime.h"
/***********************************************************************
 *  This code is part of the Small Scale Parallel Programming Assignment
 *
 * SS Assignment
 * Author:  Simeon FEREZ S392371
 * Date:    February-2023
 ***********************************************************************/

#include <iostream>
#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers
#include "../csr.h"

// ----------------------- Global variables ----------------------- //

#define XBD 64
#define YBD 16
const dim3 BLOCK_DIM(XBD,YBD);
int ntimes = 20;

// ----------------------- Function prototypes ----------------------- //

// Simple CPU implementation of matrix multiplication.
void CpuMatrixMultiply(const csr* A, const double* B, double*  C)
{
// Multiply the matrix A by the vector B by CSR format and store the result in C
    for (int i = 0; i < A->M; i++) {
        double t = 0;
        for (int j = A->IRP[i]; j < A->IRP[i+1]; j++) {
            t += B[A->JA[j]] * A->AS[j];
        }
        C[i] = t;
    }
}

__device__ void rowReduce(volatile double *sdata, int tid, int s) {
    switch(s){
        case 16:  sdata[tid] += sdata[tid + 16];
        case  8:  sdata[tid] += sdata[tid +  8];
        case  4:  sdata[tid] += sdata[tid +  4];
        case  2:  sdata[tid] += sdata[tid +  2];
        case  1:  sdata[tid] += sdata[tid +  1];
    }
}

// GPU implementation of matrix_vector product 2D block 1D grid
__global__ void gpuMatrixVector(const int *IRP, const int *JA, const double* AS, const double* B, double*  C, int M) {

    __shared__ double ax[YBD][XBD];
    int tr     = threadIdx.y;
    int tc     = threadIdx.x;
    int row    = blockIdx.x*blockDim.y + tr;
    int offset = IRP[row];
    int end    = IRP[row+1];
    int i      = 0;

    int s;
    ax[tr][tc] = 0.0;

    if (row < M) {
        for (i = offset + tc; i < (end-XBD); i += XBD*2) {
            ax[tr][tc] += AS[i] * B[JA[i]] + AS[i+XBD] * B[JA[i+XBD]];
        }
        for (; i < end; i += XBD) {
            ax[tr][tc] += AS[i] * B[JA[i]];
        }
    }
    __syncthreads();
    // Reduce the row
    for (s=XBD/2; s >=32; s >>=1){
        if (tc<s)
            ax[tr][tc] += ax[tr][tc+s];
        __syncthreads();
    }

    s = min(16,XBD/2);
    if (tc < s) rowReduce(&(ax[tr][0]),tc,s);
    // Write the result
    if (tc == 0) {
        C[row] = ax[tr][0];
    }
}

int main(int argc, char** argv) {

    if (argc < 2) {
        fprintf(stderr,"Usage: %s  filename\n",argv[0]);
    }

    csr* A = new csr(argv[1]);

    // ----------------------- Host memory initialisation ----------------------- //

    int nrowsc = A->M;
    int ncolsc = A->N;

    double* h_B = (double*) malloc(ncolsc * sizeof(double));
    double* h_C = (double*) malloc(nrowsc * sizeof(double));
    double *h_C_proof = (double*) malloc(nrowsc * sizeof(double));

    // ----------------------- Matrix Initialization  ----------------------- //

    for (int i = 0; i < ncolsc; i++) {
        h_B[i] = 1.0;
    }
    for (int i = 0; i < nrowsc; i++) {
        h_C[i] = 0.0;
    }
    for (int i = 0; i < nrowsc; i++) {
        h_C_proof[i] = 0.0;
    }

    // ---------------------- Device memory initialisation ---------------------- //

    int *d_JA, *d_IRP;
    double *d_AS, *d_B, *d_C;

    checkCudaErrors(hipMalloc((void**) &d_B, ncolsc * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_C, nrowsc * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_JA, A->nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &d_IRP, (A->M+1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &d_AS, A->nz * sizeof(double)));

    // Copy matrices from the host (CPU) to the device (GPU).
    checkCudaErrors(hipMemcpy(d_JA, A->JA, A->nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_IRP, A->IRP, (A->M+1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_AS, A->AS, A->nz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, ncolsc * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C, h_C, nrowsc * sizeof(double), hipMemcpyHostToDevice));

    // ------------------------ Calculations on the CPU ------------------------- //

    CpuMatrixMultiply(A,h_B,h_C_proof);

    // ------------------------ Calculations on the GPU ------------------------- //

    dim3 GRID_DIM((nrowsc + BLOCK_DIM.y - 1)/BLOCK_DIM.y, 1, 1);

    double bestTime = 1e30;
    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);
    for (int i = 0; i < ntimes; i++) {
        timer->reset();
        timer->start();
        gpuMatrixVector<<<GRID_DIM, BLOCK_DIM >>> (d_IRP, d_JA, d_AS, d_B, d_C, nrowsc);
        hipDeviceSynchronize();
        timer->stop();
        bestTime = min(bestTime, timer->getTime());
    }

    float gpuflops = 2.e-6*A->nz/ timer->getTime();

    // ----------------------- Check results ----------------------- //

    // Download the resulting vector d_y from the device and store it in h_y_d.
    checkCudaErrors(hipMemcpy(h_C, d_C, nrowsc * sizeof(double), hipMemcpyDeviceToHost));
    //find max error
    double max_error = 0;
    for (int i = 0; i < nrowsc; i++) {
        double error = h_C[i] - h_C_proof[i];
        if (error < 0) error = -error;
        if (error > max_error) {max_error = error;}
    }

    // ------------------------------- Print results ------------------------------ //

    std::cout << "CSR CUDA 2DB 1DG Unroll 2H\t Time " << bestTime << " ms \t GFLOPS " << gpuflops << " \t Max error " << max_error << "\t Block size " << XBD << "x" << YBD << "Grid size " << GRID_DIM.x << "x" << GRID_DIM.y << std::endl;

    // ------------------------------- Cleaning up ------------------------------ //

    delete timer;
    // Free the device memory.
    checkCudaErrors(hipFree(d_JA));
    checkCudaErrors(hipFree(d_AS));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_IRP));

    // Free the host memory.

    free(h_B);
    free(h_C);
    free(h_C_proof);

    A->~csr();

    return 0;
}
