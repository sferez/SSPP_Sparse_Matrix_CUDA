#include "hip/hip_runtime.h"
/***********************************************************************
 *  This code is part of the Small Scale Parallel Programming Assignment
 *
 * SS Assignment
 * Author:  Simeon FEREZ S392371
 * Date:    February-2023
 ***********************************************************************/

#include <iostream>
#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers
#include "../ellpack.h"

// ----------------------- Global variables ----------------------- //

#define XBD 64
const dim3 BLOCK_DIM(XBD);
int ntimes = 20;

// ----------------------- Function prototypes ----------------------- //

// Simple CPU implementation of matrix multiplication.
void CpuMatrixMultiply(const ellpack* A, const double* B, double*  C)
{
    for (int i = 0; i < A->M; i++) {
        double t0 = 0.0;
        for (int j = 0; j < A->max_nz; j++) {
            if (A->JA[i][j] == -1) break;
            t0 += A->AS[i][j] * B[A->JA[i][j]];
        }
        C[i] = t0;
    }
}

__device__ void rowReduce(volatile double *sdata, int tid, int s) {
    switch(s){
        case 16:  sdata[tid] += sdata[tid + 16];
        case  8:  sdata[tid] += sdata[tid +  8];
        case  4:  sdata[tid] += sdata[tid +  4];
        case  2:  sdata[tid] += sdata[tid +  2];
        case  1:  sdata[tid] += sdata[tid +  1];
    }
}

// GPU implementation of matrix_vector product 1D block 1D grid
__global__ void gpuMatrixVector(const int *JA, const double* AS, const double* B, double*  C, int M, int max_nz) {

    //create shared memory
    __shared__ double sdata[XBD];
    int tc     = threadIdx.x;
    int row    = blockIdx.x;

    int s;
    sdata[tc] = 0.0;

    if (row < M) {
        // Starting address of indexing within matrix A
        int idxm = row * max_nz + tc;
        int ic   = tc;
        double t  = 0.0;
        for ( ; ic < max_nz; ic += XBD) {
            if (JA[idxm] == -1) break;
            t += AS[idxm] * B[JA[idxm]];
            idxm += XBD;
        }
        sdata[tc] = t;
    }
    __syncthreads();

    //reduce
    for (s = XBD/2; s >= 32; s >>= 1) {
        if (tc < s) {
            sdata[tc] += sdata[tc + s];
        }
        __syncthreads();
    }
    s = min(16,XBD/2);
    if (tc < s) rowReduce(sdata, tc, s);
    if (tc == 0) C[row] = sdata[0];
}

int main(int argc, char** argv) {

    if (argc < 2) {
        fprintf(stderr,"Usage: %s  filename\n",argv[0]);
    }

    ellpack* A = new ellpack(argv[1]);

    // ----------------------- Host memory initialisation ----------------------- //

    int nrowsc = A->M;
    int ncolsc = A->N;

    double* h_B = (double*) malloc(ncolsc * sizeof(double));
    double* h_C = (double*) malloc(nrowsc * sizeof(double));
    double *h_C_proof = (double*) malloc(nrowsc * sizeof(double));

    // ----------------------- Matrix Initialization  ----------------------- //

    for (int i = 0; i < ncolsc; i++) {
        h_B[i] = 1.0;
    }
    for (int i = 0; i < nrowsc; i++) {
        h_C[i] = 0.0;
    }
    for (int i = 0; i < nrowsc; i++) {
        h_C_proof[i] = 0.0;
    }

    //convert JA and AS to 1D arrays
    int *h_JA = (int*) malloc(nrowsc * A->max_nz * sizeof(int));
    double *h_AS = (double*) malloc(nrowsc * A->max_nz * sizeof(double));
    for (int i = 0; i < nrowsc; i++) {
        for (int j = 0; j < A->max_nz; j++) {
            h_JA[i*A->max_nz + j] = A->JA[i][j];
            h_AS[i*A->max_nz + j] = A->AS[i][j];
        }
    }

    // ---------------------- Device memory initialisation ---------------------- //

    int *d_JA;
    double *d_AS, *d_B, *d_C;

    checkCudaErrors(hipMalloc((void**) &d_JA, nrowsc * A->max_nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &d_AS, nrowsc * A->max_nz * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_B, ncolsc * sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_C, nrowsc * sizeof(double)));

    // Copy matrices from the host (CPU) to the device (GPU).
    checkCudaErrors(hipMemcpy(d_JA, h_JA, nrowsc * A->max_nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_AS, h_AS, nrowsc * A->max_nz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, ncolsc * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C, h_C, nrowsc * sizeof(double), hipMemcpyHostToDevice));

    // ------------------------ Calculations on the CPU ------------------------- //

    CpuMatrixMultiply(A,h_B,h_C_proof);

    // ------------------------ Calculations on the GPU ------------------------- //

    dim3 GRID_DIM((nrowsc + BLOCK_DIM.y - 1)/BLOCK_DIM.y, 1, 1);

    double bestTime = 1e30;
    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);
    for (int i = 0; i < ntimes; i++) {
        timer->reset();
        timer->start();
        gpuMatrixVector<<<GRID_DIM, BLOCK_DIM >>> (d_JA, d_AS, d_B, d_C, nrowsc, A->max_nz);
        hipDeviceSynchronize();
        timer->stop();
        bestTime = min(bestTime, timer->getTime());
    }

    float gpuflops = 2.e-6*A->nz/ timer->getTime();

    // ----------------------- Check results ----------------------- //

    // Download the resulting vector d_y from the device and store it in h_y_d.
    checkCudaErrors(hipMemcpy(h_C, d_C, nrowsc * sizeof(double), hipMemcpyDeviceToHost));
    //find max error
    double max_error = 0;
    for (int i = 0; i < nrowsc; i++) {
        double error = h_C[i] - h_C_proof[i];
        if (error < 0) error = -error;
        if (error > max_error) {max_error = error;}
    }

    // ------------------------------- Print results ------------------------------ //

    std::cout << "ELLPACK CUDA 1DB 1DG \t\t Time " << bestTime << " ms \t GFLOPS " << gpuflops << " \t Max error " << max_error << "\t Block size " << XBD << "x1 " << "Grid size " << GRID_DIM.x << "x" << GRID_DIM.y << std::endl;

    // ------------------------------- Cleaning up ------------------------------ //

    delete timer;
    // Free the device memory.
    checkCudaErrors(hipFree(d_JA));
    checkCudaErrors(hipFree(d_AS));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    // Free the host memory.
    free(h_JA);
    free(h_AS);
    free(h_B);
    free(h_C);
    free(h_C_proof);

    A->~ellpack();

    return 0;
}
